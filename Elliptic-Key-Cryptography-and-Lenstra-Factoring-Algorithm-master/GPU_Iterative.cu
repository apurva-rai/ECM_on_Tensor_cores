
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <algorithm>
#include <stdlib.h>
#include <time.h>

using namespace std;

__device__
int xgcd(int a, int b, int *x, int *y)
{

  int prevx = 1, x1 = 0, prevy = 0, y1 = 1;

  while (b)
  {

    int q = a/b;

    prevx = x1;
    x1 = prevx - q*x1;
    prevy = y1;
    y1 = prevy - q*y1;
    b = a % b;
    a = b;

  }

  *x = prevx;
  *y = prevy;

  return a;

}

__global__
void modinv(int *x, int *y, int n)
{

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
  {

    int a, b;
    int g = xgcd(x[i], y[i], &a, &b);

    if (g != 1) return;
    else (a%y[i] + y[i]) % y[i];

  }
}

int main(void)
{
  int N = 1<<22;
  int *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(int));
  hipMallocManaged(&y, N*sizeof(int));

  srand((unsigned)time(NULL));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = rand()%10+1;
    y[i] = 11;

  }

  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  modinv<<<numBlocks, blockSize>>>(x, y, N);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Free memory
  hipFree(x);
  hipFree(y);

  return 0;
}
